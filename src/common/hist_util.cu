#include "hip/hip_runtime.h"
/*!
 * Copyright 2018 XGBoost contributors
 */

#include <xgboost/logging.h>

#include <thrust/copy.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>

#include <utility>
#include <vector>
#include <memory>
#include <mutex>

#include "hist_util.h"
#include "xgboost/host_device_vector.h"
#include "device_helpers.cuh"
#include "quantile.h"
#include "../tree/param.h"
#include "../data/adapter.h"
#include "math.h"
#include "../data/device_adapter.cuh"

namespace xgboost {
namespace common {

using WQSketch = DenseCuts::WQSketch;

__global__ void FindCutsK(WQSketch::Entry* __restrict__ cuts,
                          const bst_float* __restrict__ data,
                          const float* __restrict__ cum_weights,
                          int nsamples,
                          int ncuts) {
  // ncuts < nsamples
  int icut = threadIdx.x + blockIdx.x * blockDim.x;
  if (icut >= ncuts) {
    return;
  }
  int isample = 0;
  if (icut == 0) {
    isample = 0;
  } else if (icut == ncuts - 1) {
    isample = nsamples - 1;
  } else {
    bst_float rank = cum_weights[nsamples - 1] / static_cast<float>(ncuts - 1)
        * static_cast<float>(icut);
    // -1 is used because cum_weights is an inclusive sum
    isample = dh::UpperBound(cum_weights, nsamples, rank);
    isample = max(0, min(isample, nsamples - 1));
  }
  // repeated values will be filtered out on the CPU
  bst_float rmin = isample > 0 ? cum_weights[isample - 1] : 0;
  bst_float rmax = cum_weights[isample];
  cuts[icut] = WQSketch::Entry(rmin, rmax, rmax - rmin, data[isample]);
}

// predictate for thrust filtering that returns true if the element is not a NaN
struct IsNotNaN {
  __device__ bool operator()(float a) const { return !isnan(a); }
};

__global__ void UnpackFeaturesK(float* __restrict__ fvalues,
                                float* __restrict__ feature_weights,
                                const size_t* __restrict__ row_ptrs,
                                const float* __restrict__ weights,
                                Entry* entries,
                                size_t nrows_array,
                                size_t row_begin_ptr,
                                size_t nrows) {
  size_t irow = threadIdx.x + size_t(blockIdx.x) * blockDim.x;
  if (irow >= nrows) {
    return;
  }
  size_t row_length = row_ptrs[irow + 1] - row_ptrs[irow];
  int icol = threadIdx.y + blockIdx.y * blockDim.y;
  if (icol >= row_length) {
    return;
  }
  Entry entry = entries[row_ptrs[irow] - row_begin_ptr + icol];
  size_t ind = entry.index * nrows_array + irow;
  // if weights are present, ensure that a non-NaN value is written to weights
  // if and only if it is also written to features
  if (!isnan(entry.fvalue) && (weights == nullptr || !isnan(weights[irow]))) {
    fvalues[ind] = entry.fvalue;
    if (feature_weights != nullptr && weights != nullptr) {
      feature_weights[ind] = weights[irow];
    }
  }
}

  using SketchEntry = WQuantileSketch<bst_float, bst_float>::Entry;
/*!
 * \brief A container that holds the device sketches across all
 *  sparse page batches which are distributed to different devices.
 *  As sketches are aggregated by column, the mutex guards
 *  multiple devices pushing sketch summary for the same column
 *  across distinct rows.
 */
struct SketchContainer {
  std::vector<DenseCuts::WQSketch> sketches_;  // NOLINT
  static constexpr int kOmpNumColsParallelizeLimit = 1000;

  SketchContainer(int max_bin,size_t num_columns,size_t num_rows )  {
    // Initialize Sketches for this dmatrix
    sketches_.resize(num_columns);
#pragma omp parallel for default(none) shared(max_bin) schedule(static) \
if (num_columns> kOmpNumColsParallelizeLimit)  // NOLINT
    for (int icol = 0; icol < num_columns; ++icol) {  // NOLINT
      sketches_[icol].Init(num_rows, 1.0 / (8 * max_bin));
    }
  }

  /**
   * \brief Pushes cuts to the sketches.
   *
   * \param entries_per_column  The entries per column.
   * \param entries             Vector of cuts from all columns, length entries_per_column * num_columns.
   * \param column_scan         Exclusive scan of column sizes. Used to detect cases where there are
   *                            fewer entries than we have storage for.
   */
  void Push(size_t entries_per_column,
            const thrust::host_vector<SketchEntry>& entries,
            const thrust::host_vector<size_t>& column_scan) {
#pragma omp parallel for default(none) schedule( \
    static) if (sketches_.size() >               \
                SketchContainer::kOmpNumColsParallelizeLimit)  // NOLINT
    for (int icol = 0; icol < sketches_.size(); ++icol) {
      size_t column_size = column_scan[icol + 1] - column_scan[icol];
      if (column_size == 0) continue;
      WQuantileSketch<bst_float, bst_float>::SummaryContainer summary;
      size_t num_available_cuts =
          std::min(size_t(entries_per_column), column_size);
      summary.Reserve(num_available_cuts);
      summary.MakeFromSorted(&entries[entries_per_column * icol],
                             num_available_cuts);

      sketches_[icol].PushSummary(summary);
    }
  }

  // Prevent copying/assigning/moving this as its internals can't be
  // assigned/copied/moved
  SketchContainer(const SketchContainer&) = delete;
  SketchContainer(const SketchContainer &&) = delete;
  SketchContainer &operator=(const SketchContainer &) = delete;
  SketchContainer &operator=(const SketchContainer &&) = delete;
};

// finds quantiles on the GPU
class GPUSketcher {
 public:
  GPUSketcher(int device, int max_bin, int gpu_nrows)
      : device_(device), max_bin_(max_bin), gpu_batch_nrows_(gpu_nrows), row_stride_(0) {}

  ~GPUSketcher() {  // NOLINT
    dh::safe_cuda(hipSetDevice(device_));
  }

  void SketchBatch(const SparsePage &batch, const MetaInfo &info) {
    n_rows_ = batch.Size();

    Init(batch, info, gpu_batch_nrows_);
    Sketch(batch, info);
    ComputeRowStride();
  }

  /* Builds the sketches on the GPU for the dmatrix and returns the row stride
   * for the entire dataset */
  size_t Sketch(DMatrix *dmat, DenseCuts *hmat) {
    const MetaInfo& info = dmat->Info();

    row_stride_ = 0;
    sketch_container_.reset(new SketchContainer(max_bin_, dmat->Info().num_col_,dmat->Info().num_row_));
    for (const auto& batch : dmat->GetBatches<SparsePage>()) {
      this->SketchBatch(batch, info);
    }

    hmat->Init(&sketch_container_->sketches_, max_bin_, info.num_row_);
    return row_stride_;
  }

  // This needs to be public because of the __device__ lambda.
  void ComputeRowStride() {
    // Find the row stride for this batch
    auto row_iter = row_ptrs_.begin();
    // Functor for finding the maximum row size for this batch
    auto get_size = [=] __device__(size_t row) {
      return row_iter[row + 1] - row_iter[row];
    }; // NOLINT

    auto counting = thrust::make_counting_iterator(size_t(0));
    using TransformT = thrust::transform_iterator<decltype(get_size), decltype(counting), size_t>;
    TransformT row_size_iter = TransformT(counting, get_size);
    size_t batch_row_stride =
        thrust::reduce(row_size_iter, row_size_iter + n_rows_, 0, thrust::maximum<size_t>());
    row_stride_ = std::max(row_stride_, batch_row_stride);
  }

  // This needs to be public because of the __device__ lambda.
  void FindColumnCuts(size_t batch_nrows, size_t icol) {
    size_t tmp_size = tmp_storage_.size();
    // filter out NaNs in feature values
    auto fvalues_begin = fvalues_.data() + icol * gpu_batch_nrows_;
    hipcub::DeviceSelect::If(tmp_storage_.data().get(),
                          tmp_size,
                          fvalues_begin,
                          fvalues_cur_.data(),
                          num_elements_.begin(),
                          batch_nrows,
                          IsNotNaN());
    size_t nfvalues_cur = 0;
    thrust::copy_n(num_elements_.begin(), 1, &nfvalues_cur);

    // compute cumulative weights using a prefix scan
    if (has_weights_) {
      // filter out NaNs in weights;
      // since hipcub::DeviceSelect::If performs stable filtering,
      // the weights are stored in the correct positions
      auto feature_weights_begin = feature_weights_.data() + icol * gpu_batch_nrows_;
      hipcub::DeviceSelect::If(tmp_storage_.data().get(),
                            tmp_size,
                            feature_weights_begin,
                            weights_.data().get(),
                            num_elements_.begin(),
                            batch_nrows,
                            IsNotNaN());

      // sort the values and weights
      hipcub::DeviceRadixSort::SortPairs(tmp_storage_.data().get(),
                                      tmp_size,
                                      fvalues_cur_.data().get(),
                                      fvalues_begin.get(),
                                      weights_.data().get(),
                                      weights2_.data().get(),
                                      nfvalues_cur);

      // sum the weights to get cumulative weight values
      hipcub::DeviceScan::InclusiveSum(tmp_storage_.data().get(),
                                    tmp_size,
                                    weights2_.begin(),
                                    weights_.begin(),
                                    nfvalues_cur);
    } else {
      // sort the batch values
      hipcub::DeviceRadixSort::SortKeys(tmp_storage_.data().get(),
                                     tmp_size,
                                     fvalues_cur_.data().get(),
                                     fvalues_begin.get(),
                                     nfvalues_cur);

      // fill in cumulative weights with counting iterator
      thrust::copy_n(thrust::make_counting_iterator(1), nfvalues_cur, weights_.begin());
    }

    // remove repeated items and sum the weights across them;
    // non-negative weights are assumed
    hipcub::DeviceReduce::ReduceByKey(tmp_storage_.data().get(),
                                   tmp_size,
                                   fvalues_begin,
                                   fvalues_cur_.begin(),
                                   weights_.begin(),
                                   weights2_.begin(),
                                   num_elements_.begin(),
                                   thrust::maximum<bst_float>(),
                                   nfvalues_cur);
    size_t n_unique = 0;
    thrust::copy_n(num_elements_.begin(), 1, &n_unique);

    // extract cuts
    n_cuts_cur_[icol] = std::min(n_cuts_, n_unique);
    // if less elements than cuts: copy all elements with their weights
    if (n_cuts_ > n_unique) {
      float* weights2_ptr = weights2_.data().get();
      float* fvalues_ptr = fvalues_cur_.data().get();
      WQSketch::Entry* cuts_ptr = cuts_d_.data().get() + icol * n_cuts_;
      dh::LaunchN(device_, n_unique, [=]__device__(size_t i) {
        bst_float rmax = weights2_ptr[i];
        bst_float rmin = i > 0 ? weights2_ptr[i - 1] : 0;
        cuts_ptr[i] = WQSketch::Entry(rmin, rmax, rmax - rmin, fvalues_ptr[i]);
      });
    } else if (n_cuts_cur_[icol] > 0) {
      // if more elements than cuts: use binary search on cumulative weights
      uint32_t constexpr kBlockThreads = 256;
      uint32_t const kGrids = common::DivRoundUp(n_cuts_cur_[icol], kBlockThreads);
      dh::LaunchKernel {kGrids, kBlockThreads} (
          FindCutsK,
          cuts_d_.data().get() + icol * n_cuts_,
          fvalues_cur_.data().get(),
          weights2_.data().get(),
          n_unique,
          n_cuts_cur_[icol]);
      dh::safe_cuda(hipGetLastError());  // NOLINT
    }
  }

 private:
  void Init(const SparsePage& row_batch, const MetaInfo& info, int gpu_batch_nrows) {
    num_cols_ = info.num_col_;
    has_weights_ = info.weights_.Size() > 0;

    // find the batch size
    if (gpu_batch_nrows == 0) {
      // By default, use no more than 1/16th of GPU memory
      gpu_batch_nrows_ = dh::TotalMemory(device_) / (16 * num_cols_ * sizeof(Entry));
    } else if (gpu_batch_nrows == -1) {
      gpu_batch_nrows_ = n_rows_;
    } else {
      gpu_batch_nrows_ = gpu_batch_nrows;
    }
    if (gpu_batch_nrows_ > n_rows_) {
      gpu_batch_nrows_ = n_rows_;
    }

    constexpr int kFactor = 8;
    double eps = 1.0 / (kFactor * max_bin_);
    size_t dummy_nlevel;
    WQSketch::LimitSizeLevel(gpu_batch_nrows_, eps, &dummy_nlevel, &n_cuts_);

    // allocate necessary GPU buffers
    dh::safe_cuda(hipSetDevice(device_));

    entries_.resize(gpu_batch_nrows_ * num_cols_);
    fvalues_.resize(gpu_batch_nrows_ * num_cols_);
    fvalues_cur_.resize(gpu_batch_nrows_);
    cuts_d_.resize(n_cuts_ * num_cols_);
    cuts_h_.resize(n_cuts_ * num_cols_);
    weights_.resize(gpu_batch_nrows_);
    weights2_.resize(gpu_batch_nrows_);
    num_elements_.resize(1);

    if (has_weights_) {
      feature_weights_.resize(gpu_batch_nrows_ * num_cols_);
    }
    n_cuts_cur_.resize(num_cols_);

    // allocate storage for CUB algorithms; the size is the maximum of the sizes
    // required for various algorithm
    size_t tmp_size = 0, cur_tmp_size = 0;
    // size for sorting
    if (has_weights_) {
      hipcub::DeviceRadixSort::SortPairs(nullptr,
                                      cur_tmp_size,
                                      fvalues_cur_.data().get(),
                                      fvalues_.data().get(),
                                      weights_.data().get(),
                                      weights2_.data().get(),
                                      gpu_batch_nrows_);
    } else {
      hipcub::DeviceRadixSort::SortKeys(nullptr,
                                     cur_tmp_size,
                                     fvalues_cur_.data().get(),
                                     fvalues_.data().get(),
                                     gpu_batch_nrows_);
    }
    tmp_size = std::max(tmp_size, cur_tmp_size);
    // size for inclusive scan
    if (has_weights_) {
      hipcub::DeviceScan::InclusiveSum(nullptr,
                                    cur_tmp_size,
                                    weights2_.begin(),
                                    weights_.begin(),
                                    gpu_batch_nrows_);
      tmp_size = std::max(tmp_size, cur_tmp_size);
    }
    // size for reduction by key
    hipcub::DeviceReduce::ReduceByKey(nullptr,
                                   cur_tmp_size,
                                   fvalues_.begin(),
                                   fvalues_cur_.begin(),
                                   weights_.begin(),
                                   weights2_.begin(),
                                   num_elements_.begin(),
                                   thrust::maximum<bst_float>(),
                                   gpu_batch_nrows_);
    tmp_size = std::max(tmp_size, cur_tmp_size);
    // size for filtering
    hipcub::DeviceSelect::If(nullptr,
                          cur_tmp_size,
                          fvalues_.begin(),
                          fvalues_cur_.begin(),
                          num_elements_.begin(),
                          gpu_batch_nrows_,
                          IsNotNaN());
    tmp_size = std::max(tmp_size, cur_tmp_size);

    tmp_storage_.resize(tmp_size);
  }

  void Sketch(const SparsePage& row_batch, const MetaInfo& info) {
    // copy rows to the device
    dh::safe_cuda(hipSetDevice(device_));
    const auto& offset_vec = row_batch.offset.HostVector();
    row_ptrs_.resize(n_rows_ + 1);
    thrust::copy(offset_vec.data(), offset_vec.data() + n_rows_ + 1, row_ptrs_.begin());
    size_t gpu_nbatches = common::DivRoundUp(n_rows_, gpu_batch_nrows_);
    for (size_t gpu_batch = 0; gpu_batch < gpu_nbatches; ++gpu_batch) {
      SketchBatch(row_batch, info, gpu_batch);
    }
  }

  void SketchBatch(const SparsePage& row_batch, const MetaInfo& info, size_t gpu_batch) {
    // compute start and end indices
    size_t batch_row_begin = gpu_batch * gpu_batch_nrows_;
    size_t batch_row_end = std::min((gpu_batch + 1) * gpu_batch_nrows_,
                                    static_cast<size_t>(n_rows_));
    size_t batch_nrows = batch_row_end - batch_row_begin;

    const auto& offset_vec = row_batch.offset.HostVector();
    const auto& data_vec = row_batch.data.HostVector();

    size_t n_entries = offset_vec[batch_row_end] - offset_vec[batch_row_begin];
    // copy the batch to the GPU
    dh::safe_cuda(hipMemcpyAsync(entries_.data().get(),
                                  data_vec.data() + offset_vec[batch_row_begin],
                                  n_entries * sizeof(Entry),
                                  hipMemcpyDefault));
    // copy the weights if necessary
    if (has_weights_) {
      const auto& weights_vec = info.weights_.HostVector();
      dh::safe_cuda(hipMemcpyAsync(weights_.data().get(),
                                    weights_vec.data() + batch_row_begin,
                                    batch_nrows * sizeof(bst_float),
                                    hipMemcpyDefault));
    }

    // unpack the features; also unpack weights if present
    thrust::fill(fvalues_.begin(), fvalues_.end(), NAN);
    if (has_weights_) {
      thrust::fill(feature_weights_.begin(), feature_weights_.end(), NAN);
    }

    dim3 block3(16, 64, 1);
    // NOTE: This will typically support ~ 4M features - 64K*64
    dim3 grid3(common::DivRoundUp(batch_nrows, block3.x),
               common::DivRoundUp(num_cols_, block3.y), 1);
    dh::LaunchKernel {grid3, block3} (
        UnpackFeaturesK,
        fvalues_.data().get(),
        has_weights_ ? feature_weights_.data().get() : nullptr,
        row_ptrs_.data().get() + batch_row_begin,
        has_weights_ ? weights_.data().get() : nullptr, entries_.data().get(),
        gpu_batch_nrows_,
        offset_vec[batch_row_begin],
        batch_nrows);

    for (int icol = 0; icol < num_cols_; ++icol) {
      FindColumnCuts(batch_nrows, icol);
    }

    // add cuts into sketches
    thrust::copy(cuts_d_.begin(), cuts_d_.end(), cuts_h_.begin());
#pragma omp parallel for default(none) schedule(static) \
if (num_cols_ > SketchContainer::kOmpNumColsParallelizeLimit) // NOLINT
    for (int icol = 0; icol < num_cols_; ++icol) {
      WQSketch::SummaryContainer summary;
      summary.Reserve(n_cuts_);
      summary.MakeFromSorted(&cuts_h_[n_cuts_ * icol], n_cuts_cur_[icol]);

      sketch_container_->sketches_[icol].PushSummary(summary);

    }
  }

  const int device_;
  const int max_bin_;
  int gpu_batch_nrows_;
  size_t row_stride_;
  std::unique_ptr<SketchContainer> sketch_container_;

  bst_uint n_rows_{};
  int num_cols_{0};
  size_t n_cuts_{0};
  bool has_weights_{false};

  dh::device_vector<size_t> row_ptrs_{};
  dh::device_vector<Entry> entries_{};
  dh::device_vector<bst_float> fvalues_{};
  dh::device_vector<bst_float> feature_weights_{};
  dh::device_vector<bst_float> fvalues_cur_{};
  dh::device_vector<WQSketch::Entry> cuts_d_{};
  thrust::host_vector<WQSketch::Entry> cuts_h_{};
  dh::device_vector<bst_float> weights_{};
  dh::device_vector<bst_float> weights2_{};
  std::vector<size_t> n_cuts_cur_{};
  dh::device_vector<size_t> num_elements_{};
  dh::device_vector<char> tmp_storage_{};
};

size_t DeviceSketchOld(int device,
                    int max_bin,
                    int gpu_batch_nrows,
                    DMatrix* dmat,
                    HistogramCuts* hmat) {
  GPUSketcher sketcher(device, max_bin, gpu_batch_nrows);
  // We only need to return the result in HistogramCuts container, so it is safe to
  // use a pointer of local HistogramCutsDense
  DenseCuts dense_cuts(hmat);
  auto res = sketcher.Sketch(dmat, &dense_cuts);
  return res;
}

struct EntryCompareOp {
  __device__ bool operator()(const Entry& a, const Entry& b) {
    if (a.index == b.index) {
      return a.fvalue < b.fvalue;
    }
    return a.index < b.index;
  }
};

/**
 * \brief Extracts the cuts from sorted data.
 *
 * \param device                The device.
 * \param cuts                  Output cuts
 * \param num_cuts_per_feature  Number of cuts per feature.
 * \param sorted_data           Sorted entries in segments of columns
 * \param column_sizes_scan     Describes the boundaries of column segments in sorted data
 */
void ExtractCuts(int device, Span<SketchEntry> cuts,
                 size_t num_cuts_per_feature, Span<Entry> sorted_data,
                 Span<size_t> column_sizes_scan) {
  dh::LaunchN(device, cuts.size(), [=] __device__(size_t idx) {
    // Each thread is responsible for obtaining one cut from the sorted input
    size_t column_idx = idx / num_cuts_per_feature;
    size_t column_size =
        column_sizes_scan[column_idx + 1] - column_sizes_scan[column_idx];
    size_t num_available_cuts =
        std::min(size_t(num_cuts_per_feature), column_size);
    size_t cut_idx = idx % num_cuts_per_feature;
    if (cut_idx >= num_available_cuts) return;

    Span<Entry> column_entries =
        sorted_data.subspan(column_sizes_scan[column_idx], column_size);

    size_t rank = (column_entries.size() * cut_idx) / num_available_cuts;
    auto value = column_entries[rank].fvalue;
    cuts[idx] = SketchEntry(rank, rank + 1, 1, value);
  });
}

void ProcessBatch(int device, const SparsePage& page, size_t begin, size_t end,
  SketchContainer *sketch_container, int num_cuts, size_t num_columns) {
  dh::XGBCachingDeviceAllocator<char> alloc;
  const auto & host_data = page.data.ConstHostVector();
  dh::device_vector<Entry> sorted_entries( host_data.begin()+begin,host_data.begin()+end);
  thrust::sort(thrust::cuda::par(alloc), sorted_entries.begin(),
    sorted_entries.end(), EntryCompareOp());
  dh::caching_device_vector<size_t> column_sizes_scan(num_columns + 1,
                                                      0);
  auto d_column_sizes_scan = column_sizes_scan.data().get();
  auto d_sorted_entries = sorted_entries.data().get();
  dh::LaunchN(device, sorted_entries.size(), [=] __device__(size_t idx)
  {
    auto& e = d_sorted_entries[idx];
      atomicAdd(reinterpret_cast<unsigned long long*>(  // NOLINT
        &d_column_sizes_scan[e.index]),
        static_cast<unsigned long long>(1));  // NOLINT
  });
  thrust::exclusive_scan(thrust::cuda::par(alloc), column_sizes_scan.begin(),
                         column_sizes_scan.end(), column_sizes_scan.begin());
  thrust::host_vector<size_t > host_column_sizes_scan(column_sizes_scan);

  dh::caching_device_vector<SketchEntry> cuts(num_columns * num_cuts);
  ExtractCuts(device, {cuts.data().get(), cuts.size()}, num_cuts,
              {sorted_entries.data().get(), sorted_entries.size()},
              {column_sizes_scan.data().get(), column_sizes_scan.size()});

  // add cuts into sketches
  thrust::host_vector<SketchEntry> host_cuts(cuts);
  sketch_container->Push(num_cuts, host_cuts, host_column_sizes_scan);
}

HistogramCuts DeviceSketch(int device, DMatrix* dmat, int max_bins,
                           size_t sketch_batch_num_elements) {

  HistogramCuts cuts;
  DenseCuts dense_cuts(&cuts);
  SketchContainer sketch_container(max_bins, dmat->Info().num_col_,
                                   dmat->Info().num_row_);

  constexpr int kFactor = 8;
  double eps = 1.0 / (kFactor * max_bins);
  size_t dummy_nlevel;
  size_t num_cuts;
  WQuantileSketch<bst_float, bst_float>::LimitSizeLevel(
       dmat->Info().num_row_, eps, &dummy_nlevel, &num_cuts);
  num_cuts = std::min(num_cuts, dmat->Info().num_row_);
  if (sketch_batch_num_elements == 0) {
    sketch_batch_num_elements = dmat->Info().num_nonzero_;
  }
  for (const auto &batch : dmat->GetBatches<SparsePage>()) {
    size_t batch_nnz = batch.data.ConstHostVector().size();
    for (auto begin = 0ull; begin < batch_nnz; begin += sketch_batch_num_elements) {
      size_t end = std::min(batch_nnz, begin + sketch_batch_num_elements);
      ProcessBatch(device, batch, begin, end, &sketch_container, num_cuts,
                   dmat->Info().num_col_);
    }
  }

  dense_cuts.Init(&sketch_container.sketches_,max_bins, dmat->Info().num_row_);
  return cuts;
}

struct IsValidFunctor : public thrust::unary_function<Entry, bool> {
  explicit IsValidFunctor(float missing) : missing(missing) {}

  float missing;
  __device__ bool operator()(const data::COOTuple& e) const {
    if (common::CheckNAN(e.value) || e.value == missing) {
      return false;
    }
    return true;
  }
  __device__ bool operator()(const Entry& e) const {
    if (common::CheckNAN(e.fvalue) || e.fvalue == missing) {
      return false;
    }
    return true;
  }
};

template <typename ReturnT, typename IterT, typename FuncT>
thrust::transform_iterator<FuncT, IterT, ReturnT> MakeTransformIterator(
    IterT iter, FuncT func) {
  return thrust::transform_iterator<FuncT, IterT, ReturnT>(iter, func);
}

template <typename AdapterT>
void ProcessBatch(AdapterT *adapter, size_t begin, size_t end, float missing,
  SketchContainer *sketch_container, int num_cuts) {
  dh::XGBCachingDeviceAllocator<char> alloc;
  adapter->BeforeFirst();
  adapter->Next();
  auto& batch = adapter->Value();
  // Enforce single batch
  CHECK(!adapter->Next());

  auto batch_iter = MakeTransformIterator<data::COOTuple>(thrust::make_counting_iterator(0llu), [=] __device__(size_t idx) {
    return batch.GetElement(idx);
  });
  auto entry_iter = MakeTransformIterator<Entry>(
    thrust::make_counting_iterator(0llu), [=] __device__(size_t idx) {
    return Entry(batch.GetElement(idx).column_idx,
      batch.GetElement(idx).value);
  });

  // Work out how many valid entries we have in each column
  dh::caching_device_vector<size_t> column_sizes_scan(adapter->NumColumns() + 1,
    0);
  auto d_column_sizes_scan = column_sizes_scan.data().get();
  IsValidFunctor is_valid(missing);
  dh::LaunchN(adapter->DeviceIdx(), end - begin, [=] __device__(size_t idx)
  {
    auto &e = batch_iter[begin + idx];
    if (is_valid(e)) {
      atomicAdd(reinterpret_cast<unsigned long long*>(  // NOLINT
        &d_column_sizes_scan[e.column_idx]),
        static_cast<unsigned long long>(1));  // NOLINT
    }
  });
  thrust::exclusive_scan(thrust::cuda::par(alloc), column_sizes_scan.begin(),
    column_sizes_scan.end(), column_sizes_scan.begin());
  thrust::host_vector<size_t > host_column_sizes_scan(column_sizes_scan);
  size_t num_valid = host_column_sizes_scan.back();

  // Copy current subset of valid elements into temporary storage and sort
  thrust::device_vector<Entry> sorted_entries(num_valid);
  thrust::copy_if(thrust::cuda::par(alloc), entry_iter + begin, entry_iter + end, sorted_entries.begin(), is_valid);
  thrust::sort(thrust::cuda::par(alloc), sorted_entries.begin(),
    sorted_entries.end(), EntryCompareOp());

  // Extract the cuts from all columns concurrently
  dh::caching_device_vector<SketchEntry> cuts(adapter->NumColumns() * num_cuts);
  ExtractCuts(adapter->DeviceIdx(), {cuts.data().get(), cuts.size()}, num_cuts,
    {sorted_entries.data().get(), sorted_entries.size()},
    {column_sizes_scan.data().get(), column_sizes_scan.size()});

  // Push cuts into sketches stored in host memory
  thrust::host_vector<SketchEntry> host_cuts(cuts);
  sketch_container->Push(num_cuts, host_cuts, host_column_sizes_scan);
}


template <typename AdapterT>
HistogramCuts AdapterDeviceSketch(AdapterT *adapter, int num_bins, float missing, size_t sketch_batch_num_elements) {

  CHECK(adapter->NumRows() != data::kAdapterUnknownSize);
  CHECK(adapter->NumColumns() != data::kAdapterUnknownSize);

  adapter->BeforeFirst();
  adapter->Next();
  auto& batch = adapter->Value();

  // Enforce single batch
  CHECK(!adapter->Next());

  HistogramCuts cuts;
  DenseCuts dense_cuts(&cuts);
  SketchContainer sketch_container(num_bins, adapter->NumColumns(),
                                   adapter->NumRows());

  constexpr int kFactor = 8;
  double eps = 1.0 / (kFactor * num_bins);
  size_t dummy_nlevel;
  size_t num_cuts;
  WQuantileSketch<bst_float, bst_float>::LimitSizeLevel(
       adapter->NumRows(), eps, &dummy_nlevel, &num_cuts);
  num_cuts = std::min(num_cuts, adapter->NumRows());
  if (sketch_batch_num_elements == 0) {
    sketch_batch_num_elements = batch.Size();
  }
  for (auto begin = 0ull; begin < batch.Size(); begin += sketch_batch_num_elements) {
    size_t end = std::min(batch.Size(), begin + sketch_batch_num_elements);
    ProcessBatch(adapter, begin, end, missing, &sketch_container, num_cuts);
  }

  dense_cuts.Init(&sketch_container.sketches_, num_bins, adapter->NumRows());
  return cuts;
}

template HistogramCuts AdapterDeviceSketch(data::CudfAdapter* adapter,
                                           int num_bins, float missing,
                                           size_t sketch_batch_size);
template HistogramCuts AdapterDeviceSketch(data::CupyAdapter* adapter,
                                           int num_bins, float missing,
                                           size_t sketch_batch_size);
}  // namespace common
}  // namespace xgboost
