#include "hip/hip_runtime.h"
// Copyright (c) 2019 by Contributors

#include <xgboost/c_api.h>
#include <memory>
#include <xgboost/data.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/scan.h>

#include "./c_api_error.h"
#include "../data/simple_csr_source.h"
#include "../common/host_device_vector.h"
#include "../common/device_helpers.cuh"



namespace xgboost {
typedef unsigned char gdf_valid_type;
typedef int gdf_size_type; /**< Limits the maximum size of a gdf_column to 2^31-1 */
typedef enum {
    GDF_invalid=0,
    GDF_INT8,
    GDF_INT16,
    GDF_INT32,
    GDF_INT64,
    GDF_FLOAT32,
    GDF_FLOAT64,
    GDF_DATE32,   	/**< int32_t days since the UNIX epoch */
    GDF_DATE64,   	/**< int64_t milliseconds since the UNIX epoch */
    GDF_TIMESTAMP,	/**< Exact timestamp encoded with int64 since UNIX epoch (Default unit millisecond) */
    GDF_CATEGORY,
    GDF_STRING,
    N_GDF_TYPES, 	/* additional types should go BEFORE N_GDF_TYPES */
} gdf_dtype;

typedef enum {
	TIME_UNIT_NONE=0, // default (undefined)
	TIME_UNIT_s,   // second
	TIME_UNIT_ms,  // millisecond
	TIME_UNIT_us,  // microsecond
	TIME_UNIT_ns   // nanosecond
} gdf_time_unit;

typedef struct {
	gdf_time_unit time_unit;
	// here we can also hold info for decimal datatype or any other datatype that requires additional information
} gdf_dtype_extra_info;

typedef struct gdf_column_{
    void *data;                       /**< Pointer to the columns data */
    gdf_valid_type *valid;            /**< Pointer to the columns validity bit mask where the 'i'th bit indicates if the 'i'th row is NULL */
    gdf_size_type size;               /**< Number of data elements in the columns data buffer. Limited to 2^31 - 1.*/
    gdf_dtype dtype;                  /**< The datatype of the column's data */
    gdf_size_type null_count;         /**< The number of NULL values in the column's data */
    gdf_dtype_extra_info dtype_info;
    char *			col_name;			// host-side:	null terminated string
} gdf_column;

struct CsrCudf {
  Entry* data;
  size_t* offsets;
  size_t n_nz;
  size_t n_rows;
  size_t n_cols;
};

void RunConverter(gdf_column** gdf_data, CsrCudf* csr);

//--- private CUDA functions / kernels
__global__ void cuda_create_csr_k
(void *cudf_data, gdf_valid_type* valid, gdf_dtype dtype, int col, Entry* data,
 gdf_size_type *offsets, size_t n_rows);

__global__ void determine_valid_rec_count_k
(gdf_valid_type* valid, size_t n_rows, size_t n_cols, size_t* offset);

__device__ int WhichBitmap(int record) { return record / 8; }
__device__ int WhichBit(int bit) { return bit % 8; }
__device__ int CheckBit(gdf_valid_type data, int bit) {
  gdf_valid_type bit_mask[8] = {1, 2, 4, 8, 16, 32, 64, 128};
  return data & bit_mask[bit];
}

__device__ bool IsValid(gdf_valid_type* valid, int tid) {
  if (valid == nullptr)
    return true;
  int bitmap_idx = WhichBitmap(tid);
  int bit_idx = WhichBit(tid);
  gdf_valid_type bitmap = valid[bitmap_idx];
  return CheckBit(bitmap, bit_idx);
}

// Convert a CUDF into a CSR CUDF
void CUDFToCSR(gdf_column** cudf_data, int n_cols, CsrCudf* csr) {
  size_t n_rows = cudf_data[0]->size;

  // the first step is to create an array that counts the number of valid entries per row
  // this is done by each thread looking across its row and checking the valid bits
  int threads = 1024;
  int blocks = (n_rows + threads - 1) / threads;

  size_t* offsets = csr->offsets;
  dh::safe_cuda(hipMemset(offsets, 0, sizeof(gdf_size_type) * (n_rows + 1)));

  if (blocks > 0) {
    for (int i = 0; i < n_cols; ++i) {
      determine_valid_rec_count_k<<<blocks, threads>>>
        (cudf_data[i]->valid, n_rows, n_cols, offsets);
      dh::safe_cuda(hipGetLastError());
      dh::safe_cuda(hipDeviceSynchronize());
    }
  }

  // compute the number of elements
  thrust::device_ptr<size_t> offsets_begin(offsets);
  int64_t n_elements = thrust::reduce
    (offsets_begin, offsets_begin + n_rows, 0ull, thrust::plus<size_t>());

  // now do an exclusive scan to compute the offsets for where to write data
  thrust::exclusive_scan(offsets_begin, offsets_begin + n_rows + 1, offsets_begin);

  csr->n_rows = n_rows;
  csr->n_cols = n_cols;
  csr->n_nz = n_elements;

  // process based on data type
  RunConverter(cudf_data, csr);
}

void RunConverter(gdf_column** cudf_data, CsrCudf* csr) {
  size_t n_cols = csr->n_cols;
  size_t n_rows = csr->n_rows;

  int threads = 256;
  int blocks = (n_rows + threads - 1) / threads;

  // temporary offsets for writing data
  thrust::device_ptr<size_t> offset_begin(csr->offsets);
  thrust::device_vector<size_t> offsets2(offset_begin, offset_begin + n_rows + 1);

  // move the data and create the CSR
  if (blocks > 0) {
    for (int col = 0; col < n_cols; ++col) {
      gdf_column *cudf = cudf_data[col];
      cuda_create_csr_k<<<blocks, threads>>>
        (cudf->data, cudf->valid, cudf->dtype, col, csr->data,
         offsets2.data().get(), n_rows);
      dh::safe_cuda(hipGetLastError());
    }
  }
}

// move data over into CSR and possibly convert the format
__global__ void cuda_create_csr_k
(void* cudf_data, gdf_valid_type* valid, gdf_dtype dtype, int col,
 Entry* data, size_t* offsets, size_t n_rows) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if (tid >= n_rows)
    return;
  gdf_size_type offset_idx = offsets[tid];
  if (IsValid(valid, tid)) {
    data[offset_idx].fvalue = ConvertDataElement(cudf_data, tid, dtype);
    data[offset_idx].index = col;
    ++offsets[tid];
  }
}

// compute the number of valid entries per row
__global__ void determine_valid_rec_count_k
(gdf_valid_type *valid, size_t n_rows, size_t n_cols, gdf_size_type* offset) {

  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if (tid >= n_rows)
    return;
  if (IsValid(valid, tid))
    ++offset[tid];
}

void InitFromCUDF(SimpleCSRSource *source, gdf_column** cols, size_t n_cols) {
  CHECK_GT(n_cols, 0);
  size_t n_rows = cols[0]->size;
  info.num_col_ = n_cols;
  info.num_row_ = n_rows;
  size_t n_entries = 0;
  for (size_t i = 0; i < n_cols; ++i) {
    CHECK_EQ(n_rows, cols[i]->size);
    n_entries += cols[i]->size - cols[i]->null_count;
  }
  info.num_nonzero_ = n_entries;
  // TODO(canonizer): use the same devices as by the rest of xgboost
  GPUSet devices = GPUSet::Range(0, 1);
  page_.offset.Reshard(GPUDistribution::Overlap(devices, 1));
  // TODO(canonizer): use the real row offsets for the multi-GPU case
  std::vector<size_t> device_offsets{0, n_entries};
  page_.data.Reshard(GPUDistribution::Explicit(devices, device_offsets));
  page_.offset.Resize(n_rows + 1);
  page_.data.Resize(n_entries);
  CsrCudf csr;
  csr.data = page_.data.DevicePointer(0);
  csr.offsets = page_.offset.DevicePointer(0);
  csr.n_nz = 0;
  csr.n_rows = n_rows;
  csr.n_cols = n_cols;
  CUDFToCSR(cols, n_cols, &csr);
}

int XGDMatrixCreateFromCUDF
(void **cols, size_t n_cols, DMatrixHandle *out) {
  API_BEGIN();
  std::unique_ptr<data::SimpleCSRSource> source(new data::SimpleCSRSource());
  InitFromCUDF(source.get(), cols, n_cols);
  *out = new std::shared_ptr<DMatrix>(DMatrix::Create(std::move(source)));
  API_END();
}


XGB_DLL int XGDMatrixSetCUDFInfo(DMatrixHandle handle,
                                const char *field,
                                void **cols,
                                size_t n_cols) {
  API_BEGIN();
  CHECK_HANDLE();
  // static_cast<std::shared_ptr<DMatrix>*>(handle)
  //  ->get()->Info().SetCUDFInfo(field, cols, n_cols);
  API_END();
}
}

