#include "hip/hip_runtime.h"
/*!
 * Copyright 2015-2019 by Contributors
 * \file regression_obj.cu
 * \brief Definition of single-value regression and classification objectives.
 * \author Tianqi Chen, Kailong Chen
 */

#include <dmlc/omp.h>
#include <xgboost/logging.h>
#include <xgboost/objective.h>
#include <cmath>
#include <memory>
#include <vector>

#if defined(__HIPCC__)
#include <thrust/binary_search.h>
#include <thrust/partition.h>
#include <thrust/sort.h>
#include <thrust/transform_reduce.h>
#include <thrust/pair.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/zip_iterator.h>
#include "../common/device_helpers.cuh"
#endif

#include <rabit/c_api.h>
#include "xgboost/host_device_vector.h"
#include "xgboost/json.h"
#include "xgboost/parameter.h"
#include "xgboost/span.h"

#include "../common/transform.h"
#include "../common/common.h"
#include "./regression_loss.h"
#include "../common/device_helpers.cuh"


namespace xgboost {
namespace obj {

#if defined(XGBOOST_USE_CUDA)
DMLC_REGISTRY_FILE_TAG(regression_obj_gpu);
#endif  // defined(XGBOOST_USE_CUDA)

struct RegLossParam : public XGBoostParameter<RegLossParam> {
  float scale_pos_weight;
  // declare parameters
  DMLC_DECLARE_PARAMETER(RegLossParam) {
    DMLC_DECLARE_FIELD(scale_pos_weight).set_default(1.0f).set_lower_bound(0.0f)
      .describe("Scale the weight of positive examples by this factor");
  }
};

template<typename Loss>
class RegLossObj : public ObjFunction {
 protected:
  HostDeviceVector<int> label_correct_;

 public:
  RegLossObj() = default;

  void Configure(const std::vector<std::pair<std::string, std::string> >& args) override {
    param_.UpdateAllowUnknown(args);
  }

  void GetGradient(const HostDeviceVector<bst_float>& preds,
                   const MetaInfo &info,
                   int iter,
                   HostDeviceVector<GradientPair>* out_gpair) override {
    if (info.labels_.Size() == 0U) {
      LOG(WARNING) << "Label set is empty.";
    }
    CHECK_EQ(preds.Size(), info.labels_.Size())
        << " " << "labels are not correctly provided"
        << "preds.size=" << preds.Size() << ", label.size=" << info.labels_.Size() << ", "
        << "Loss: " << Loss::Name();
    size_t const ndata = preds.Size();
    out_gpair->Resize(ndata);
    auto device = tparam_->gpu_id;
    label_correct_.Resize(1);
    label_correct_.Fill(1);

    bool is_null_weight = info.weights_.Size() == 0;
    if (!is_null_weight) {
      CHECK_EQ(info.weights_.Size(), ndata)
          << "Number of weights should be equal to number of data points.";
    }
    auto scale_pos_weight = param_.scale_pos_weight;
    common::Transform<>::Init(
        [=] XGBOOST_DEVICE(size_t _idx,
                           common::Span<int> _label_correct,
                           common::Span<GradientPair> _out_gpair,
                           common::Span<const bst_float> _preds,
                           common::Span<const bst_float> _labels,
                           common::Span<const bst_float> _weights) {
          bst_float p = Loss::PredTransform(_preds[_idx]);
          bst_float w = is_null_weight ? 1.0f : _weights[_idx];
          bst_float label = _labels[_idx];
          if (label == 1.0f) {
            w *= scale_pos_weight;
          }
          if (!Loss::CheckLabel(label)) {
            // If there is an incorrect label, the host code will know.
            _label_correct[0] = 0;
          }
          _out_gpair[_idx] = GradientPair(Loss::FirstOrderGradient(p, label) * w,
                                          Loss::SecondOrderGradient(p, label) * w);
        },
        common::Range{0, static_cast<int64_t>(ndata)}, device).Eval(
            &label_correct_, out_gpair, &preds, &info.labels_, &info.weights_);

    // copy "label correct" flags back to host
    std::vector<int>& label_correct_h = label_correct_.HostVector();
    for (auto const flag : label_correct_h) {
      if (flag == 0) {
        LOG(FATAL) << Loss::LabelErrorMsg();
      }
    }
  }

 public:
  const char* DefaultEvalMetric() const override {
    return Loss::DefaultEvalMetric();
  }

  void PredTransform(HostDeviceVector<float> *io_preds) override {
    common::Transform<>::Init(
        [] XGBOOST_DEVICE(size_t _idx, common::Span<float> _preds) {
          _preds[_idx] = Loss::PredTransform(_preds[_idx]);
        }, common::Range{0, static_cast<int64_t>(io_preds->Size())},
        tparam_->gpu_id)
        .Eval(io_preds);
  }

  float ProbToMargin(float base_score) const override {
    return Loss::ProbToMargin(base_score);
  }

  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String(Loss::Name());
    out["reg_loss_param"] = toJson(param_);
  }

  void LoadConfig(Json const& in) override {
    fromJson(in["reg_loss_param"], &param_);
  }

 protected:
  RegLossParam param_;
};

// register the objective functions
DMLC_REGISTER_PARAMETER(RegLossParam);

XGBOOST_REGISTER_OBJECTIVE(SquaredLossRegression, LinearSquareLoss::Name())
.describe("Regression with squared error.")
.set_body([]() { return new RegLossObj<LinearSquareLoss>(); });

XGBOOST_REGISTER_OBJECTIVE(SquareLogError, SquaredLogError::Name())
.describe("Regression with root mean squared logarithmic error.")
.set_body([]() { return new RegLossObj<SquaredLogError>(); });

XGBOOST_REGISTER_OBJECTIVE(LogisticRegression, LogisticRegression::Name())
.describe("Logistic regression for probability regression task.")
.set_body([]() { return new RegLossObj<LogisticRegression>(); });

XGBOOST_REGISTER_OBJECTIVE(LogisticClassification, LogisticClassification::Name())
.describe("Logistic regression for binary classification task.")
.set_body([]() { return new RegLossObj<LogisticClassification>(); });

XGBOOST_REGISTER_OBJECTIVE(LogisticRaw, LogisticRaw::Name())
.describe("Logistic regression for classification, output score "
          "before logistic transformation.")
.set_body([]() { return new RegLossObj<LogisticRaw>(); });

// Deprecated functions
XGBOOST_REGISTER_OBJECTIVE(LinearRegression, "reg:linear")
.describe("Regression with squared error.")
.set_body([]() {
    LOG(WARNING) << "reg:linear is now deprecated in favor of reg:squarederror.";
    return new RegLossObj<LinearSquareLoss>(); });
// End deprecated

// declare parameter
struct PoissonRegressionParam : public XGBoostParameter<PoissonRegressionParam> {
  float max_delta_step;
  DMLC_DECLARE_PARAMETER(PoissonRegressionParam) {
    DMLC_DECLARE_FIELD(max_delta_step).set_lower_bound(0.0f).set_default(0.7f)
        .describe("Maximum delta step we allow each weight estimation to be." \
                  " This parameter is required for possion regression.");
  }
};

// poisson regression for count
class PoissonRegression : public ObjFunction {
 public:
  // declare functions
  void Configure(const std::vector<std::pair<std::string, std::string> >& args) override {
    param_.UpdateAllowUnknown(args);
  }

  void GetGradient(const HostDeviceVector<bst_float>& preds,
                   const MetaInfo &info,
                   int iter,
                   HostDeviceVector<GradientPair> *out_gpair) override {
    CHECK_NE(info.labels_.Size(), 0U) << "label set cannot be empty";
    CHECK_EQ(preds.Size(), info.labels_.Size()) << "labels are not correctly provided";
    size_t const ndata = preds.Size();
    out_gpair->Resize(ndata);
    auto device = tparam_->gpu_id;
    label_correct_.Resize(1);
    label_correct_.Fill(1);

    bool is_null_weight = info.weights_.Size() == 0;
    if (!is_null_weight) {
      CHECK_EQ(info.weights_.Size(), ndata)
          << "Number of weights should be equal to number of data points.";
    }
    bst_float max_delta_step = param_.max_delta_step;
    common::Transform<>::Init(
        [=] XGBOOST_DEVICE(size_t _idx,
                           common::Span<int> _label_correct,
                           common::Span<GradientPair> _out_gpair,
                           common::Span<const bst_float> _preds,
                           common::Span<const bst_float> _labels,
                           common::Span<const bst_float> _weights) {
          bst_float p = _preds[_idx];
          bst_float w = is_null_weight ? 1.0f : _weights[_idx];
          bst_float y = _labels[_idx];
          if (y < 0.0f) {
            _label_correct[0] = 0;
          }
          _out_gpair[_idx] = GradientPair{(expf(p) - y) * w,
                                          expf(p + max_delta_step) * w};
        },
        common::Range{0, static_cast<int64_t>(ndata)}, device).Eval(
            &label_correct_, out_gpair, &preds, &info.labels_, &info.weights_);
    // copy "label correct" flags back to host
    std::vector<int>& label_correct_h = label_correct_.HostVector();
    for (auto const flag : label_correct_h) {
      if (flag == 0) {
        LOG(FATAL) << "PoissonRegression: label must be nonnegative";
      }
    }
  }
  void PredTransform(HostDeviceVector<bst_float> *io_preds) override {
    common::Transform<>::Init(
        [] XGBOOST_DEVICE(size_t _idx, common::Span<bst_float> _preds) {
          _preds[_idx] = expf(_preds[_idx]);
        },
        common::Range{0, static_cast<int64_t>(io_preds->Size())},
        tparam_->gpu_id)
        .Eval(io_preds);
  }
  void EvalTransform(HostDeviceVector<bst_float> *io_preds) override {
    PredTransform(io_preds);
  }
  bst_float ProbToMargin(bst_float base_score) const override {
    return std::log(base_score);
  }
  const char* DefaultEvalMetric() const override {
    return "poisson-nloglik";
  }

  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String("count:poisson");
    out["poisson_regression_param"] = toJson(param_);
  }

  void LoadConfig(Json const& in) override {
    fromJson(in["poisson_regression_param"], &param_);
  }

 private:
  PoissonRegressionParam param_;
  HostDeviceVector<int> label_correct_;
};

// register the objective functions
DMLC_REGISTER_PARAMETER(PoissonRegressionParam);

XGBOOST_REGISTER_OBJECTIVE(PoissonRegression, "count:poisson")
.describe("Possion regression for count data.")
.set_body([]() { return new PoissonRegression(); });


// cox regression for survival data (negative values mean they are censored)
class CoxRegression : public ObjFunction {
 public:
  void Configure(
      const std::vector<std::pair<std::string, std::string> > &args) override {}

  void GetGradient(const HostDeviceVector<bst_float>& preds,
                   const MetaInfo &info,
                   int iter,
                   HostDeviceVector<GradientPair> *out_gpair) override {
    CHECK_NE(info.labels_.Size(), 0U) << "label set cannot be empty";
    CHECK_EQ(preds.Size(), info.labels_.Size()) << "labels are not correctly provided";
    const auto& preds_h = preds.HostVector();
    out_gpair->Resize(preds_h.size());
    auto& gpair = out_gpair->HostVector();
    const std::vector<size_t> &label_order = info.LabelAbsSort();

    const omp_ulong ndata = static_cast<omp_ulong>(preds_h.size()); // NOLINT(*)
    const bool is_null_weight = info.weights_.Size() == 0;
    if (!is_null_weight) {
      CHECK_EQ(info.weights_.Size(), ndata)
          << "Number of weights should be equal to number of data points.";
    }

    // pre-compute a sum
    double exp_p_sum = 0;  // we use double because we might need the precision with large datasets
    for (omp_ulong i = 0; i < ndata; ++i) {
      exp_p_sum += std::exp(preds_h[label_order[i]]);
    }

    // start calculating grad and hess
    const auto& labels = info.labels_.HostVector();
    double r_k = 0;
    double s_k = 0;
    double last_exp_p = 0.0;
    double last_abs_y = 0.0;
    double accumulated_sum = 0;
    for (omp_ulong i = 0; i < ndata; ++i) { // NOLINT(*)
      const size_t ind = label_order[i];
      const double p = preds_h[ind];
      const double exp_p = std::exp(p);
      const double w = info.GetWeight(ind);
      const double y = labels[ind];
      const double abs_y = std::abs(y);

      // only update the denominator after we move forward in time (labels are sorted)
      // this is Breslow's method for ties
      accumulated_sum += last_exp_p;
      if (last_abs_y < abs_y) {
        exp_p_sum -= accumulated_sum;
        accumulated_sum = 0;
      } else {
        CHECK(last_abs_y <= abs_y) << "CoxRegression: labels must be in sorted order, " <<
                                      "MetaInfo::LabelArgsort failed!";
      }

      if (y > 0) {
        r_k += 1.0/exp_p_sum;
        s_k += 1.0/(exp_p_sum*exp_p_sum);
      }

      const double grad = exp_p*r_k - static_cast<bst_float>(y > 0);
      const double hess = exp_p*r_k - exp_p*exp_p * s_k;
      gpair.at(ind) = GradientPair(grad * w, hess * w);

      last_abs_y = abs_y;
      last_exp_p = exp_p;
    }
  }
  void PredTransform(HostDeviceVector<bst_float> *io_preds) override {
    std::vector<bst_float> &preds = io_preds->HostVector();
    const long ndata = static_cast<long>(preds.size()); // NOLINT(*)
#pragma omp parallel for schedule(static)
    for (long j = 0; j < ndata; ++j) {  // NOLINT(*)
      preds[j] = std::exp(preds[j]);
    }
  }
  void EvalTransform(HostDeviceVector<bst_float> *io_preds) override {
    PredTransform(io_preds);
  }
  bst_float ProbToMargin(bst_float base_score) const override {
    return std::log(base_score);
  }
  const char* DefaultEvalMetric() const override {
    return "cox-nloglik";
  }

  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String("survival:cox");
  }
  void LoadConfig(Json const&) override {}
};

// register the objective function
XGBOOST_REGISTER_OBJECTIVE(CoxRegression, "survival:cox")
.describe("Cox regression for censored survival data (negative labels are considered censored).")
.set_body([]() { return new CoxRegression(); });

// gamma regression
class GammaRegression : public ObjFunction {
 public:
  void Configure(
      const std::vector<std::pair<std::string, std::string> > &args) override {}

  void GetGradient(const HostDeviceVector<bst_float> &preds,
                   const MetaInfo &info,
                   int iter,
                   HostDeviceVector<GradientPair> *out_gpair) override {
    CHECK_NE(info.labels_.Size(), 0U) << "label set cannot be empty";
    CHECK_EQ(preds.Size(), info.labels_.Size()) << "labels are not correctly provided";
    const size_t ndata = preds.Size();
    auto device = tparam_->gpu_id;
    out_gpair->Resize(ndata);
    label_correct_.Resize(1);
    label_correct_.Fill(1);

    const bool is_null_weight = info.weights_.Size() == 0;
    if (!is_null_weight) {
      CHECK_EQ(info.weights_.Size(), ndata)
          << "Number of weights should be equal to number of data points.";
    }
    common::Transform<>::Init(
        [=] XGBOOST_DEVICE(size_t _idx,
                           common::Span<int> _label_correct,
                           common::Span<GradientPair> _out_gpair,
                           common::Span<const bst_float> _preds,
                           common::Span<const bst_float> _labels,
                           common::Span<const bst_float> _weights) {
          bst_float p = _preds[_idx];
          bst_float w = is_null_weight ? 1.0f : _weights[_idx];
          bst_float y = _labels[_idx];
          if (y < 0.0f) {
            _label_correct[0] = 0;
          }
          _out_gpair[_idx] = GradientPair((1 - y / expf(p)) * w, y / expf(p) * w);
        },
        common::Range{0, static_cast<int64_t>(ndata)}, device).Eval(
            &label_correct_, out_gpair, &preds, &info.labels_, &info.weights_);

    // copy "label correct" flags back to host
    std::vector<int>& label_correct_h = label_correct_.HostVector();
    for (auto const flag : label_correct_h) {
      if (flag == 0) {
        LOG(FATAL) << "GammaRegression: label must be nonnegative";
      }
    }
  }
  void PredTransform(HostDeviceVector<bst_float> *io_preds) override {
    common::Transform<>::Init(
        [] XGBOOST_DEVICE(size_t _idx, common::Span<bst_float> _preds) {
          _preds[_idx] = expf(_preds[_idx]);
        },
        common::Range{0, static_cast<int64_t>(io_preds->Size())},
        tparam_->gpu_id)
        .Eval(io_preds);
  }
  void EvalTransform(HostDeviceVector<bst_float> *io_preds) override {
    PredTransform(io_preds);
  }
  bst_float ProbToMargin(bst_float base_score) const override {
    return std::log(base_score);
  }
  const char* DefaultEvalMetric() const override {
    return "gamma-nloglik";
  }
  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String("reg:gamma");
  }
  void LoadConfig(Json const&) override {}

 private:
  HostDeviceVector<int> label_correct_;
};

// register the objective functions
XGBOOST_REGISTER_OBJECTIVE(GammaRegression, "reg:gamma")
.describe("Gamma regression for severity data.")
.set_body([]() { return new GammaRegression(); });


// declare parameter
struct TweedieRegressionParam : public XGBoostParameter<TweedieRegressionParam> {
  float tweedie_variance_power;
  DMLC_DECLARE_PARAMETER(TweedieRegressionParam) {
    DMLC_DECLARE_FIELD(tweedie_variance_power).set_range(1.0f, 2.0f).set_default(1.5f)
      .describe("Tweedie variance power.  Must be between in range [1, 2).");
  }
};

// tweedie regression
class TweedieRegression : public ObjFunction {
 public:
  // declare functions
  void Configure(const std::vector<std::pair<std::string, std::string> >& args) override {
    param_.UpdateAllowUnknown(args);
    std::ostringstream os;
    os << "tweedie-nloglik@" << param_.tweedie_variance_power;
    metric_ = os.str();
  }

  void GetGradient(const HostDeviceVector<bst_float>& preds,
                   const MetaInfo &info,
                   int iter,
                   HostDeviceVector<GradientPair> *out_gpair) override {
    CHECK_NE(info.labels_.Size(), 0U) << "label set cannot be empty";
    CHECK_EQ(preds.Size(), info.labels_.Size()) << "labels are not correctly provided";
    const size_t ndata = preds.Size();
    out_gpair->Resize(ndata);

    auto device = tparam_->gpu_id;
    label_correct_.Resize(1);
    label_correct_.Fill(1);

    const bool is_null_weight = info.weights_.Size() == 0;
    if (!is_null_weight) {
      CHECK_EQ(info.weights_.Size(), ndata)
          << "Number of weights should be equal to number of data points.";
    }

    const float rho = param_.tweedie_variance_power;
    common::Transform<>::Init(
        [=] XGBOOST_DEVICE(size_t _idx,
                           common::Span<int> _label_correct,
                           common::Span<GradientPair> _out_gpair,
                           common::Span<const bst_float> _preds,
                           common::Span<const bst_float> _labels,
                           common::Span<const bst_float> _weights) {
          bst_float p = _preds[_idx];
          bst_float w = is_null_weight ? 1.0f : _weights[_idx];
          bst_float y = _labels[_idx];
          if (y < 0.0f) {
            _label_correct[0] = 0;
          }
          bst_float grad = -y * expf((1 - rho) * p) + expf((2 - rho) * p);
          bst_float hess =
              -y * (1 - rho) * \
              std::exp((1 - rho) * p) + (2 - rho) * expf((2 - rho) * p);
          _out_gpair[_idx] = GradientPair(grad * w, hess * w);
        },
        common::Range{0, static_cast<int64_t>(ndata), 1}, device)
        .Eval(&label_correct_, out_gpair, &preds, &info.labels_, &info.weights_);

    // copy "label correct" flags back to host
    std::vector<int>& label_correct_h = label_correct_.HostVector();
    for (auto const flag : label_correct_h) {
      if (flag == 0) {
        LOG(FATAL) << "TweedieRegression: label must be nonnegative";
      }
    }
  }
  void PredTransform(HostDeviceVector<bst_float> *io_preds) override {
    common::Transform<>::Init(
        [] XGBOOST_DEVICE(size_t _idx, common::Span<bst_float> _preds) {
          _preds[_idx] = expf(_preds[_idx]);
        },
        common::Range{0, static_cast<int64_t>(io_preds->Size())},
        tparam_->gpu_id)
        .Eval(io_preds);
  }

  bst_float ProbToMargin(bst_float base_score) const override {
    return std::log(base_score);
  }

  const char* DefaultEvalMetric() const override {
    return metric_.c_str();
  }

  void SaveConfig(Json* p_out) const override {
    auto& out = *p_out;
    out["name"] = String("reg:tweedie");
    out["tweedie_regression_param"] = toJson(param_);
  }
  void LoadConfig(Json const& in) override {
    fromJson(in["tweedie_regression_param"], &param_);
  }

 private:
  std::string metric_;
  TweedieRegressionParam param_;
  HostDeviceVector<int> label_correct_;
};

// register the objective functions
DMLC_REGISTER_PARAMETER(TweedieRegressionParam);

XGBOOST_REGISTER_OBJECTIVE(TweedieRegression, "reg:tweedie")
.describe("Tweedie regression for insurance data.")
.set_body([]() { return new TweedieRegression(); });


class AUCExponentialObj : public ObjFunction {
 public:
  void Configure(
      const std::vector<std::pair<std::string, std::string> >& args) override {}

  std::pair<double, double> ReduceCPU(const HostDeviceVector<bst_float>& preds,
                                      const MetaInfo& info) {
    size_t const ndata = preds.Size();
    double sum_exp_pos = 0.0;
    double sum_exp_neg = 0.0;
    const auto& label = info.labels_.ConstHostVector();
    const auto& pred = preds.ConstHostVector();
#pragma omp parallel for reduction(+: sum_exp_pos, sum_exp_neg ) schedule(static)
    for (omp_ulong i = 0; i < ndata; ++i) {
      if (label[i] == 1.0) {
        sum_exp_pos += std::exp(-pred[i]);
      } else if (label[i] == 0.0) {
        sum_exp_neg += std::exp(pred[i]);
      }
    }
    return {sum_exp_pos, sum_exp_neg};
  }


#if defined(__HIPCC__)
  std::pair<double, double> ReduceGPU(const HostDeviceVector<bst_float>& preds,
                                      const MetaInfo& info) {
    preds.SetDevice(tparam_->gpu_id);
    info.labels_.SetDevice(tparam_->gpu_id);
    auto input = thrust::make_zip_iterator(thrust::make_tuple(
        preds.ConstDevicePointer(), info.labels_.ConstDevicePointer()));

    auto unary =
        [=] __device__(
            thrust::tuple<float, float> x) -> thrust::pair<double, double> {
      float p = x.get<0>();
      float y = x.get<1>();
      if (y == 1.0) {
        return {std::exp(-p), 0};
      } else {
        return {0, std::exp(p)};
      }
    };
    dh::XGBCachingDeviceAllocator<char> alloc;
    auto result = thrust::transform_reduce(
        thrust::cuda::par(alloc), input, input + preds.Size(), unary,
        thrust::pair<double, double>(0, 0),
        [=] __device__(thrust::pair<double, double> a,
                       thrust::pair<double, double> b) {
          b.first += a.first;
          b.second += a.second;
          return b;
        });
    return std::pair<float, float>(result.first, result.second);
  }
#else
  std::pair<double, double> ReduceGPU(const HostDeviceVector<bst_float>& preds,
                                      const MetaInfo& info) {
    LOG(FATAL) << "XGBoost not complied with GPU support.";
    return {0.0, 0.0};
  }
#endif

  void GetGradient(const HostDeviceVector<bst_float>& preds,
                   const MetaInfo& info, int iter,
                   HostDeviceVector<GradientPair>* out_gpair) override {
    if (info.labels_.Size() == 0U) {
      LOG(WARNING) << "Label set is empty.";
    }
    CHECK_EQ(preds.Size(), info.labels_.Size());
    out_gpair->Resize(preds.Size());

    std::pair<double, double> pos_neg_sums;
    if (tparam_->gpu_id >= 0) {
      pos_neg_sums = ReduceGPU(preds, info);
    } else {
      pos_neg_sums = ReduceCPU(preds, info);
    }
    rabit::Allreduce<rabit::op::Sum, double>(
        reinterpret_cast<double*>(&pos_neg_sums), 2);
    const bool is_null_weight = info.weights_.Size() == 0;
    label_correct_.Fill(1);

    common::Transform<>::Init(
        [=] XGBOOST_DEVICE(size_t _idx, common::Span<int> _label_correct,
                           common::Span<GradientPair> _out_gpair,
                           common::Span<const bst_float> _preds,
                           common::Span<const bst_float> _labels,
                           common::Span<const bst_float> _weights) {
          bst_float p = _preds[_idx];
          bst_float w = is_null_weight ? 1.0f : _weights[_idx];
          bst_float y = _labels[_idx];
          bst_float g = 0.0f;
          bst_float h = 0.0f;
          if (y == 1.0) {
            g = -std::exp(-p) * pos_neg_sums.second * w;
            h = -g;
          } else if (y == 0.0) {
            g = std::exp(p) * pos_neg_sums.first * w;
            h = g;
          } else {
            _label_correct[0] = 0;
          }
          _out_gpair[_idx] = GradientPair(g, h);
        },
        common::Range{0, static_cast<int64_t>(preds.Size())}, tparam_->gpu_id)
        .Eval(&label_correct_, out_gpair, &preds, &info.labels_,
              &info.weights_);

    if (label_correct_.ConstHostVector()[0] == 0) {
      LOG(FATAL) << "Label must be 0.0 or 1.0";
    }
  }

  void SaveConfig(Json* p_out) const override {}

  void LoadConfig(Json const& in) override {}

  const char* DefaultEvalMetric() const override { return "auc"; }

 private:
  HostDeviceVector<int> label_correct_{1};
};
XGBOOST_REGISTER_OBJECTIVE(AUCExponentialObj, "reg:auc_exp")
    .describe("Direct AUC optimisation with exponential surrogate function.")
    .set_body([]() { return new AUCExponentialObj(); });

class AUCSquaredObj : public ObjFunction {
 public:
  void Configure(
      const std::vector<std::pair<std::string, std::string> >& args) override {}

  std::pair<double, double> ReduceCPU(const HostDeviceVector<bst_float>& preds,
                                      const MetaInfo& info) {
    size_t const ndata = preds.Size();
    double sum_pos = 0.0;
    double sum_neg = 0.0;
    const auto& label = info.labels_.ConstHostVector();
    const auto& pred = preds.ConstHostVector();
#pragma omp parallel for reduction(+ : sum_pos, sum_neg) schedule(static)
    for (omp_ulong i = 0; i < ndata; ++i) {
      if (label[i] == 1.0) {
        sum_pos += pred[i];
      } else if (label[i] == 0.0) {
        sum_neg += pred[i];
      }
    }
    return {sum_pos, sum_neg};
  }

#if defined(__HIPCC__)
  std::pair<double, double> ReduceGPU(const HostDeviceVector<bst_float>& preds,
                                      const MetaInfo& info) {
    preds.SetDevice(tparam_->gpu_id);
    info.labels_.SetDevice(tparam_->gpu_id);
    auto input = thrust::make_zip_iterator(thrust::make_tuple(
        preds.ConstDevicePointer(), info.labels_.ConstDevicePointer()));

    auto unary =
        [=] __device__(
            thrust::tuple<float, float> x) -> thrust::pair<double, double> {
      float p = x.get<0>();
      float y = x.get<1>();
      if (y == 1.0) {
        return {p, 0};
      } else {
        return {0, p};
      }
    };
    dh::XGBCachingDeviceAllocator<char> alloc;
    auto result = thrust::transform_reduce(
        thrust::cuda::par(alloc), input, input + preds.Size(), unary,
        thrust::pair<double, double>(0, 0),
        [=] __device__(thrust::pair<double, double> a,
                       thrust::pair<double, double> b) {
          b.first += a.first;
          b.second += a.second;
          return b;
        });
    return std::pair<float, float>(result.first, result.second);
  }
#else
  std::pair<double, double> ReduceGPU(const HostDeviceVector<bst_float>& preds,
                                      const MetaInfo& info) {
    LOG(FATAL) << "XGBoost not complied with GPU support.";
    return {0.0, 0.0};
  }
#endif
  void LazyCountLabels(const MetaInfo& info) {
    // Lazily count positive/negative labels
    if (num_pos_neg_.first == 0) {
      for (auto y : info.labels_.ConstHostVector()) {
        if (y == 1.0) {
          num_pos_neg_.first += 1;
        } else {
          num_pos_neg_.second += 1;
        }
      }
      rabit::Allreduce<rabit::op::Sum, size_t>(
          reinterpret_cast<size_t*>(&num_pos_neg_), 2);
      CHECK(num_pos_neg_.first && num_pos_neg_.second)
          << "Can't have all positive or all negative labels";
    }
  }

  void GetGradient(const HostDeviceVector<bst_float>& preds,
                   const MetaInfo& info, int iter,
                   HostDeviceVector<GradientPair>* out_gpair) override {
    if (info.labels_.Size() == 0U) {
      LOG(WARNING) << "Label set is empty.";
    }
    CHECK_EQ(preds.Size(), info.labels_.Size());
    out_gpair->Resize(preds.Size());

    this->LazyCountLabels(info);
    std::pair<double, double> pos_neg_sums;
    if (tparam_->gpu_id >= 0) {
      pos_neg_sums = ReduceGPU(preds, info);
    } else {
      pos_neg_sums = ReduceCPU(preds, info);
    }
    rabit::Allreduce<rabit::op::Sum, double>(
        reinterpret_cast<double*>(&pos_neg_sums), 2);

    const bool is_null_weight = info.weights_.Size() == 0;
    label_correct_.Fill(1);
    size_t num_positive = num_pos_neg_.first;
    size_t num_negative = num_pos_neg_.second;

    common::Transform<>::Init(
        [=] XGBOOST_DEVICE(size_t _idx, common::Span<int> _label_correct,
                           common::Span<GradientPair> _out_gpair,
                           common::Span<const bst_float> _preds,
                           common::Span<const bst_float> _labels,
                           common::Span<const bst_float> _weights) {
          bst_float p = _preds[_idx];
          bst_float w = is_null_weight ? 1.0f : _weights[_idx];
          bst_float y = _labels[_idx];
          bst_float g = 0.0f;
          bst_float h = 0.0f;
          if (y == 1.0) {
            g = ((p - 1) / num_positive) -
                (pos_neg_sums.second / (num_positive * num_negative));
            h = 1.0 / num_positive;
          } else if (y == 0.0) {
            g = ((p + 1) / num_negative) -
                (pos_neg_sums.first / (num_positive * num_negative));
            h = 1.0 / num_negative;
          } else {
            _label_correct[0] = 0;
          }
          // Keep Hessian around 2 otherwise min_child_weight prevents tree
          // growth
          float normalisation = num_negative + num_positive;
          _out_gpair[_idx] =
              GradientPair(g * w * normalisation, h * w * normalisation);
        },
        common::Range{0, static_cast<int64_t>(preds.Size())}, tparam_->gpu_id)
        .Eval(&label_correct_, out_gpair, &preds, &info.labels_,
              &info.weights_);

    if (label_correct_.ConstHostVector()[0] == 0) {
      LOG(FATAL) << "Label must be 0.0 or 1.0";
    }
  }

  void SaveConfig(Json* p_out) const override {}

  void LoadConfig(Json const& in) override {}

  const char* DefaultEvalMetric() const override { return "auc"; }

 private:
  HostDeviceVector<int> label_correct_{1};
  std::pair<size_t, size_t> num_pos_neg_{0, 0};
};
XGBOOST_REGISTER_OBJECTIVE(AUCSquaredObj, "reg:auc_squared")
    .describe("Direct AUC optimisation with squared surrogate function.")
    .set_body([]() { return new AUCSquaredObj(); });

class AUCHingeObj : public ObjFunction {
 public:
  void Configure(
      const std::vector<std::pair<std::string, std::string> >& args) override {
    CHECK_EQ(rabit::GetWorldSize(), 1) << "Objective for single machine only.";
  }
#if defined(__HIPCC__)
  void GetGradientDevice(const HostDeviceVector<bst_float>& preds,
                         const MetaInfo& info, int iter,
                         HostDeviceVector<GradientPair>* out_gpair) {
    preds.SetDevice(tparam_->gpu_id);
    info.labels_.SetDevice(tparam_->gpu_id);
    info.weights_.SetDevice(tparam_->gpu_id);
    out_gpair->SetDevice(tparam_->gpu_id);
    dh::XGBCachingDeviceAllocator<char> alloc;
    dh::caching_device_vector<float> sorted_preds(
        thrust::device_pointer_cast(preds.ConstDevicePointer()),
        thrust::device_pointer_cast(preds.ConstDevicePointer() + preds.Size()));
    auto neg_iter =
        thrust::partition(thrust::cuda::par(alloc), sorted_preds.begin(),
                          sorted_preds.end(), info.labels_.ConstDevicePointer(),
                          [=] __device__(float y) { return y == 1.0; });
    size_t num_pos = neg_iter - sorted_preds.begin();
    size_t num_neg = preds.Size() - num_pos;
    // sort positives
    thrust::sort(thrust::cuda::par(alloc), sorted_preds.begin(),
                 sorted_preds.begin() + num_pos);
    // sort negatives
    thrust::sort(thrust::cuda::par(alloc), sorted_preds.begin() + num_pos,
                 sorted_preds.end());
    auto d_label = info.labels_.ConstDevicePointer();
    auto d_preds = preds.ConstDevicePointer();
    auto d_out_gpair = out_gpair->DevicePointer();
    const bool is_null_weight = info.weights_.Size() == 0;
    auto d_weight = info.weights_.ConstDevicePointer();
    common::Span<float> sorted_pos(sorted_preds.data().get(), num_pos);
    common::Span<float> sorted_neg(sorted_preds.data().get() + num_pos,
                                   num_neg);
    dh::LaunchN(tparam_->gpu_id, preds.Size(), [=] __device__(size_t idx) {
      float w = is_null_weight ? 1.0f : d_weight[idx];
      float y = d_label[idx];
      float p = d_preds[idx];
      float g, h;
      if (y == 1.0) {
        // Number of negative predictions greater than this prediction - 1
        auto itr = thrust::upper_bound(thrust::seq, sorted_neg.begin(),
                                       sorted_neg.end(), p - 1.0f);
        g = -(sorted_neg.end() - itr);
        h = 1.0;
      } else {
        // Number of positive predictions less than this prediction + 1
        auto itr = thrust::lower_bound(thrust::seq, sorted_pos.begin(),
                                       sorted_pos.end(), p + 1.0f);
        g = itr - sorted_pos.begin();
        h = 1.0;
      }
      d_out_gpair[idx] = GradientPair(g * w, h * w);
    });
  }
#else
  void GetGradientDevice(const HostDeviceVector<bst_float>& preds,
                         const MetaInfo& info, int iter,
                         HostDeviceVector<GradientPair>* out_gpair) {
    LOG(FATAL) << "XGBoost not complied with GPU support.";
  }
#endif
  void GetGradientHost(const HostDeviceVector<bst_float>& preds,
                       const MetaInfo& info, int iter,
                       HostDeviceVector<GradientPair>* out_gpair) {
    std::vector<float> sorted_pos;
    sorted_pos.reserve(preds.Size());
    std::vector<float> sorted_neg;
    sorted_neg.reserve(preds.Size());
    auto& label = info.labels_.ConstHostVector();
    auto& pred = preds.ConstHostVector();
    for (auto i = 0ull; i < preds.Size(); i++) {
      if (label[i] == 1.0) {
        sorted_pos.push_back(pred[i]);
      } else if (label[i] == 0.0) {
        sorted_neg.push_back(pred[i]);
      } else {
        LOG(FATAL) << "Label must be 0.0 or 1.0";
      }
    }
    std::sort(sorted_pos.begin(), sorted_pos.end());
    std::sort(sorted_neg.begin(), sorted_neg.end());

    size_t const ndata = preds.Size();
    auto& out = out_gpair->HostVector();
#pragma omp parallel for schedule(static)
    for (omp_ulong i = 0; i < ndata; ++i) {
      float w = info.GetWeight(i);
      float y = label[i];
      float p = pred[i];
      float g, h;
      if (y == 1.0) {
        // Number of negative predictions greater than this prediction - 1
        auto itr =
            std::upper_bound(sorted_neg.begin(), sorted_neg.end(), p - 1.0f);
        g = -(sorted_neg.end() - itr);
        h = 1.0;
      } else {
        // Number of positive predictions less than this prediction + 1
        auto itr =
            std::lower_bound(sorted_pos.begin(), sorted_pos.end(), p + 1.0f);
        g = itr - sorted_pos.begin();
        h = 1.0;
      }
      out[i] = GradientPair(g * w, h * w);
    }
  }
  void GetGradient(const HostDeviceVector<bst_float>& preds,
                   const MetaInfo& info, int iter,
                   HostDeviceVector<GradientPair>* out_gpair) override {
    if (info.labels_.Size() == 0U) {
      LOG(WARNING) << "Label set is empty.";
    }
    CHECK_EQ(preds.Size(), info.labels_.Size());
    out_gpair->Resize(preds.Size());

    if (tparam_->gpu_id >= 0) {
      this->GetGradientDevice(preds, info, iter, out_gpair);
    } else {
      this->GetGradientHost(preds, info, iter, out_gpair);
    }
  }

  void SaveConfig(Json* p_out) const override {}

  void LoadConfig(Json const& in) override {}

  const char* DefaultEvalMetric() const override { return "auc"; }

 private:
  HostDeviceVector<int> label_correct_{1};
  std::pair<size_t, size_t> num_pos_neg_{0, 0};
};
XGBOOST_REGISTER_OBJECTIVE(AUCHingeObj, "reg:auc_hinge")
    .describe("Direct AUC optimisation with hinge surrogate function.")
    .set_body([]() { return new AUCHingeObj(); });

}  // namespace obj
}  // namespace xgboost
